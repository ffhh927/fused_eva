#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <stdio.h>
#include <vector>
#include <torch/extension.h>
using data_type = float;
using namespace std;

#define index(i, j, ld) ((i) * (ld) + (j))

// cuda API error checking
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                     \
        }                                                                                          \
    } while (0)

// cublas API error checking
#define CUBLAS_CHECK(err)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (err);                                                               \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
        }                                                                                          \
    } 


__global__ void showGPU(data_type *ptr, int size){
    for (int i = 0; i < size; i++)
        printf("%-3.1f ", ptr[i]);
    printf("\n");
    
}

void printTensor(at::Tensor tensor) {
    auto sizes = tensor.sizes();
    auto data_accessor = tensor.accessor<float, 2>(); // Assuming tensor is 2D
    for (int i = 0; i < sizes[0]; i++) {
        for (int j = 0; j < sizes[1]; j++) {
            printf("%f ", data_accessor[i][j]);
        }
        printf("\n");
    }
}
void printMatrix(const std::vector<at::Tensor>& vectorOfTensors) {
    for (size_t i = 0; i < vectorOfTensors.size(); i++) {
        printf("Tensor %zu:\n", i);
        printTensor(vectorOfTensors[i]);
        printf("\n");
    }
}

std::vector<at::Tensor>& multi_group(
                const std::vector<at::Tensor>& A,
                const std::vector<at::Tensor>& B,
                std::vector<at::Tensor>& C){
   //断言
   int batchCount = A.size();
   int* m = new int[batchCount];
   int* n = new int[batchCount];
   int* k = new int[batchCount];
   data_type* alpha = new data_type[batchCount]; 
   data_type* beta = new data_type[batchCount]; 
   int* lda = new int[batchCount];
   int* ldb = new int[batchCount];
   int* ldc = new int[batchCount];
   int* group_size = new int[batchCount];
   hipblasOperation_t* transa_array = new hipblasOperation_t[batchCount];
   hipblasOperation_t* transb_array = new hipblasOperation_t[batchCount];
   
   for(int i = 0; i < batchCount; i++){
       m[i] = A[i].sizes()[0];
       n[i] = B[i].sizes()[1];
       k[i] = A[i].sizes()[1]; 
       alpha[i] = 1.0;
       beta[i] = 0.0;
       lda[i] = m[i];
       ldb[i] = k[i];
       ldc[i] = m[i]; 
       group_size[i] = 1;
       transa_array[i] = HIPBLAS_OP_N;
       transb_array[i] = HIPBLAS_OP_N;
   }
   
   // const int m = A[0].sizes()[0];
   // const int n = B[0].sizes()[1];
   // const int k = A[0].sizes()[1];
   // int batchCount = A.size();
    // 打印tensor数据
  //  printf("A\n");
    //printMatrix(A);
  //  printf("B\n");
    //printMatrix(B);

   
    std::vector<data_type*> d_A(batchCount, nullptr);
    std::vector<data_type*> d_B(batchCount, nullptr);
    std::vector<data_type*> d_C(batchCount, nullptr);

    data_type **d_A_array = nullptr, **d_B_array = nullptr, **d_C_array = nullptr;

    // 传输数据
    for (int i = 0; i < batchCount; i++) {
        d_A[i] = A[i].contiguous().data_ptr<data_type>();
        d_B[i] = B[i].contiguous().data_ptr<data_type>();
        d_C[i] = C[i].contiguous().data_ptr<data_type>();
        }
  //  printf("test00\n");
    // 分配 array
    CUDA_CHECK(hipMalloc(&d_A_array, sizeof(data_type*) * batchCount));
    CUDA_CHECK(hipMalloc(&d_B_array, sizeof(data_type*) * batchCount));
    CUDA_CHECK(hipMalloc(&d_C_array, sizeof(data_type*) * batchCount));

    // 传输数据
    CUDA_CHECK(hipMemcpy(d_A_array, d_A.data(), sizeof(data_type*) * batchCount, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_array, d_B.data(), sizeof(data_type*) * batchCount, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_array, d_C.data(), sizeof(data_type*) * batchCount, hipMemcpyHostToDevice));
       
 //   printf("test1\n");

    // 矩阵计算
    hipblasHandle_t handle;
    hipblasStatus_t status;
    status = hipblasCreate(&handle);

    cublasSgemmGroupedBatched(handle, transa_array, transb_array,//非转置
                     m, n, k,
                     alpha,
                     d_A_array, lda,
                     d_B_array, ldb,
                     beta,
                     d_C_array, ldc,
                     batchCount,
                     group_size);
 //   status =hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,//非转置
  //                          m, n, k, 
   //                         &alpha, 
    //                        d_A_array, lda,
     //                       d_B_array, ldb, 
      //                      &beta, 
       //                     d_C_array, ldc, 
        //                    batchCount);
  // hipblasSgemmBatched();
   //cublasSgemmGroupedBatched();
  //  printf("test2\n");
    if (status != HIPBLAS_STATUS_SUCCESS){
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        exit(1);
    }
    hipDeviceSynchronize();
  //  printf("test3\n");

   // printf("test4\n");
    // 释放空间
    hipblasDestroy(handle);
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    CUDA_CHECK(hipFree(d_C_array));

  //  printf("Result: \n");
    //printMatrix(C);
return C;
}
